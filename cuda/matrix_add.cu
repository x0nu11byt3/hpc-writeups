#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N_THREADS 10
#define THREADS_PER_BLOCK 2
#define N_BLOCKS 5

/* Kernel definition matrix add */
__global__ void matrix_add(float a[N_THREADS][N_THREADS], float b[N_THREADS][N_THREADS], float c[N_THREADS][N_THREADS]){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if ( i < N_THREADS && j < N_THREADS)
        c[i][j] = a[i][j] + b[i][j];
}

/* Fill matrix with random values  */
void fill_matrix_random(float **matrix, int min, int max){
    time_t seconds = (time(NULL))/3600;
    srand(seconds);

    for (int i = 0; i < N_THREADS; i++  ) {
        for (int j = 0; j < N_THREADS; j++  ) {
            float random = ((rand() % (max - min + 1))+min);
            matrix[i][j] = random;
        }
    }
}

int main(int argc, char** argv) {

    /* Init host code */
    size_t size = N_THREADS * N_THREADS * sizeof(float);

    /* Allocate input matrix host_a & host_b in host memory (RAM) */

    float *host_a = (float*)malloc(size);
    float *host_b = (float*)malloc(size);
    float *host_c = (float*)malloc(size);

    /* Inicialize input matrix */
    fill_matrix_random(&host_a, 1, 99);
    fill_matrix_random(&host_b, 1, 99);

    /* Allocate matrix in device memory (VRAM) */
    float *device_a, *device_b, *device_c;
    hipMalloc((void**)&device_a, size);
    hipMalloc((void**)&device_b, size);
    hipMalloc((void**)&device_c, size);

    /* Copy input matrix from host memory to device memory */
    hipMemcpy(device_a, host_a, size, hipMemcpyHostToDevice);
    hipMemcpy(device_b, host_b, size, hipMemcpyHostToDevice);

    /*Invoke Kernel */
    dim3 threadsPerBlock(N_THREADS,N_THREADS);
    dim3 numBlocks( N_THREADS / threadsPerBlock.x, N_THREADS / threadsPerBlock.y  );
    matrix_add<<<numBlocks,threadsPerBlock>>>(device_a,device_b,device_c);

    /* Copy result from device memory to host memory */
    hipMemcpy(host_c,device_c, size,hipMemcpyDeviceToHost);

    for (int i = 0; i < N_THREADS; i++  ){
        for (int j = 0; j < N_THREADS; j++  )
            printf("%f +  %f = %f \n" ,host_a[i][j],host_b[i][j], host_c[i][j]);
        printf("\n");
    }

    /* Free host memory */
    free(host_a);
    free(host_b);
    free(host_c);

    /* Free device memory */
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    return 0;
}
