#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N_THREADS 10
#define THREADS_PER_BLOCK 2
#define N_BLOCKS 5

/* Kernel definition vector add */
__global__ void matrix_add(float a[N_THREADS][N_THREADS], float b[N_THREADS][N_THREADS], float c[N_THREADS][N_THREADS]){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if ( i < N_THREADS && j < N_THREADS)
        c[i][j] = a[i][j] + b[i][j];
}

/* Fill matrix with random values  */
void fill_matrix_random(float **matrix, int min, int max){
    time_t seconds = (time(NULL))/3600;
    srand(seconds);

    for (int i = 0; i < N_THREADS; i++  ) {
        for (int j = 0; j < N_THREADS; j++  ) {
            float random = ((rand() % (max - min + 1))+min);
            matrix[i][j] = random;
        }
    }
}

int main(int argc, char** argv) {

    /* Init host code */
    size_t size = N_THREADS * sizeof(float);

    /* Allocate input vectors host_a & host_b in host memory (RAM) */

    float *host_a = (float*)malloc(N_THREADS * size);
    float *host_b = (float*)malloc(N_THREADS * size);
    float *host_c = (float*)malloc(N_THREADS * size);

    /* Inicialize input vectors */
    fill_matrix_random(&host_a, 1, 99);
    fill_matrix_random(&host_b, 1, 99);

    /* Allocate vectors in device memory (VRAM) */
    float *device_a, *device_b, *device_c;
    hipMalloc((void**)&device_a, N_THREADS * size);
    hipMalloc((void**)&device_b, N_THREADS * size);
    hipMalloc((void**)&device_c, N_THREADS * size);

    /* Copy input vectors from host memory to device memory */
    hipMemcpy(device_a, host_a, N_THREADS * size, hipMemcpyHostToDevice);
    hipMemcpy(device_b, host_b, N_THREADS * size, hipMemcpyHostToDevice);

    /*Invoke Kernel */
    dim3 threadsPerBlock(N_THREADS,N_THREADS);
    dim3 numBlocks( N_THREADS / threadsPerBlock.x, N_THREADS / threadsPerBlock.y  );
    matrix_add<<<numBlocks,threadsPerBlock>>>(device_a,device_b,device_c);
    //vector_add<<<1,N_THREADS>>>(device_a,device_b,device_c);

    /* Copy result from device memory to host memory */
    hipMemcpy(host_c,device_c, N_THREADS * size,hipMemcpyDeviceToHost);

    for (int i = 0; i < N_THREADS; i++  ){
        for (int j = 0; j < N_THREADS; j++  )
            printf("%f +  %f = %f \n" ,host_a[i][j],host_b[i][j], host_c[i][j]);
        printf("\n");
    }

    /* Free host memory */
    free(host_a);
    free(host_b);
    free(host_c);

    /* Free device memory */
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    return 0;
}
