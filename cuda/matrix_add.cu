#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define N_THREADS 10
#define THREADS_PER_BLOCK 2
#define N_BLOCKS 5

/* Kernel definition vector add */
__global__ void matrix_add(float a[N_THREADS][N_THREADS], float b[N_THREADS][N_THREADS], float c[N_THREADS][N_THREADS]){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if ( i < N_THREADS && j < N_THREADS)
        c[i][j] = a[i][j] + b[i][j];
}

/* Fill matrix with random values  */
void fill_matrix_random(float **matrix, int min, int max){
    time_t seconds = (time(NULL))/3600;
    srand(seconds);

    for (int i = 0; i < N_THREADS; i++  ) {
        for (int j = 0; j < N_THREADS; j++  ) {
            float random = ((rand() % (max - min + 1))+min);
            matrix[i][j] = random;
        }
    }
}

int main() {

    /* Init host code */
    size_t size = N_THREADS * sizeof(float);

    /* Allocate input vectors host_a & host_b in host memory (RAM) */
    //float *host_a = (float*)malloc(size);
    //float *host_b = (float*)malloc(size);
    //float *host_c = (float*)malloc(size);

    /* Inicialize input vectors */
    //fill_matrix_random(host_a, 1, 99);
    //fill_matrix_random(host_b, 1, 99);

    /* Allocate vectors in device memory (VRAM) */
    //float *device_a, *device_b, *device_c;
    //cudaMalloc(&device_a,size);
    //cudaMalloc(&device_b,size);
    //cudaMalloc(&device_c,size);

    /* Copy input vectors from host memory to device memory */
    //cudaMemcpy(device_a,host_a,size,cudaMemcpyHostToDevice);
    //cudaMemcpy(device_b,host_b,size,cudaMemcpyHostToDevice);

    /*Invoke Kernel */
    //dim3 threadsPerBlock(16,16);
    //dim3 numBlocks( N_THREADS / threadsPerBlock.x, N_THREADS / threadsPerBlock.y  );
    //matrix_add<<<numBlocks,threadsPerBlock>>>(device_a,device_b,device_c);
    //vector_add<<<1,N_THREADS>>>(device_a,device_b,device_c);

    /* Copy result from device memory to host memory */
    //cudaMemcpy(host_c,device_c,size,cudaMemcpyDeviceToHost);

    //for (int i = 0; i < N_THREADS; i++  )
    //    printf(" total: %f \n" , host_c[i]);

    /* Free device memory */
    //cudaFree(device_a);
    //cudaFree(device_b);
    //cudaFree(device_c);

    return 0;
}
