#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define N_THREADS 4

/* kernel definition vector add */
__global__ void vector_add(float *a, float *b, float *c){
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void fill_array_random(float *array, int min, int max){
    time_t seconds = (time(NULL))/3600;
    srand(seconds);

    for (int i = 0; i < N_THREADS; i++  ) {
        float random = ((rand() % (max - min + 1))+min);
        array[i] = random;
    }
}

int main() {

    /* Init host code */
    size_t size = N_THREADS * sizeof(float);

    /* Allocate input vectors host_a & host_b in host memory (RAM) */
    float *host_a = (float*)malloc(size);
    float *host_b = (float*)malloc(size);
    float *host_c = (float*)malloc(size);

    /* Inicialize input vectors */
    fill_array_random(host_a, 1, 99);
    fill_array_random(host_b, 1, 99);

    /* Allocate vectors in device memory (VRAM) */
    float *device_a, *device_b, *device_c;
    hipMalloc(&device_a,size);
    hipMalloc(&device_b,size);
    hipMalloc(&device_c,size);

    /* Copy input vectors from host memory to device memory */
    hipMemcpy(device_a,host_a,size,hipMemcpyHostToDevice);
    hipMemcpy(device_b,host_b,size,hipMemcpyHostToDevice);

    /*Invoke Kernel */
    vector_add<<<1,N_THREADS>>>(device_a,device_b,device_c);

    /* Copy result from device memory to host memory */
    hipMemcpy(host_c,device_c,size,hipMemcpyDeviceToHost);

    for (int i = 0; i < N_THREADS; i++  )
        printf(" total: %f \n" , host_c[i]);

    /* Free device memory */
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    return 0;
}
