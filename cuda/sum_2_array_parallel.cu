#include <stdio.h>
#include <hip/hip_runtime.h>


#define N_THREADS 2

/* kernel definition vector add */
__global__ void vector_add(float *a, float *b, float *c){
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main() {

    /* Init host code */
    size_t size = N_THREADS * sizeof(float);

    /* Allocate input vectors host_a & host_b in host memory (RAM) */
    float *host_a = (float*)malloc(size);
    float *host_b = (float*)malloc(size);
    float *host_c = (float*)malloc(size);

    /* Allocate vectors in device memory (VRAM) */
    float *device_a, *device_b, *device_c;
    hipMalloc(&device_a,size);
    hipMalloc(&device_b,size);
    hipMalloc(&device_c,size);

    /* Copy result from device memory to host memory */
    hipMemcpy(device_a,host_a,size,hipMemcpyHostToDevice);
    hipMemcpy(device_b,host_b,size,hipMemcpyHostToDevice);

    /*Invoke Kernel */
    vector_add<<<1,N_THREADS>>>(device_a,device_b,device_c);

    /* Free device memory */
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    return 0;
}
