#include <__clang_cuda_builtin_vars.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define N_THREADS 10
#define THREADS_PER_BLOCK 2
#define N_BLOCKS 5

/* Kernel definition vector add */
__global__ void vector_add(float *a, float *b, float *c){
    int i =  blockDim.x * blockIdx.x + threadIdx.x;
    while ( i < N_THREADS ) {
        c[i] = a[i] + b[i];
        i += blockDim.x;
    }
}

/* Fill array with random values  */
void fill_array_random(float *array, int min, int max){
    time_t seconds = (time(NULL))/3600;
    srand(seconds);
    for (int i = 0; i < N_THREADS; i++  ) {
        float random = ((rand() % (max - min + 1))+min);
        array[i] = random;
    }
}

int main(int argc, char** argv) {

    /* Init host code */
    size_t size = N_THREADS * sizeof(float);

    /* Allocate input vectors host_a & host_b in host memory (RAM) */
    float *host_a = (float*)malloc(size);
    float *host_b = (float*)malloc(size);
    float *host_c = (float*)malloc(size);

    /* Inicialize input vectors */
    fill_array_random(host_a, 1, 99);
    fill_array_random(host_b, 1, 99);

    /* Allocate vectors in device memory (VRAM) */
    float *device_a, *device_b, *device_c;
    hipMalloc( (void**)&device_a, size );
    hipMalloc( (void**)&device_b, size );
    hipMalloc( (void**)&device_c, size );

    /* Copy input vectors from host memory to device memory */
    hipMemcpy(device_a,host_a,size,hipMemcpyHostToDevice);
    hipMemcpy(device_b,host_b,size,hipMemcpyHostToDevice);

    /*Invoke Kernel */
    //int threads_per_block = 256;
    //int blocks_per_grid = (N_THREADS + threads_per_block - 1) / threads_per_block;
    //vector_add<<<blocks_per_grid,threads_per_block>>>(device_a,device_b,device_c);
    //vector_add<<<1,N_THREADS>>>(device_a,device_b,device_c);
    vector_add<<<N_BLOCKS,N_THREADS>>>(device_a,device_b,device_c);
    /* Copy result from device memory to host memory */
    hipMemcpy(host_c,device_c,size,hipMemcpyDeviceToHost);

    for (int i = 0; i < N_THREADS; i++  )
        printf("%f +  %f = %f \n" ,host_a[i],host_b[i], host_c[i]);

    /* Free host memory */
    free(host_a);
    free(host_b);
    free(host_c);

    /* Free device memory */
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    return 0;
}
