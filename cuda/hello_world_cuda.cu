#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void cuda_hello_world(){
    printf("[+] Hello World from GPU!\n");
}

int main() {
    cuda_hello_world<<<1,1>>>();
    return 0;
}
