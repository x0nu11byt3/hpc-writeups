#include <stdio.h>
#include <hip/hip_runtime.h>


#define N_THREADS 1

// kernel definition hello World
__global__ void cuda_hello_world(){
    printf("[+] Hello World from GPU!\n");
}

int main(int argc, char** argv) {

    // kernel invocation with N threads
    cuda_hello_world<<<1,N_THREADS>>>();

    return 0;
}
